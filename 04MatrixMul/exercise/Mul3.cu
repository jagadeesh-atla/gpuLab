/**
 *Write a CUDA program to demonstrate the followings
 1. Allocate Device Memory
 2. Transfer Data(Matrices A, B and C) from host to device
 3. Find the Product of three matrices A*B*C using 2D grid
 4. Transfer result from device to host
 5. Print the result in matrix format
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1 << 10;

__global__ void MatrixMul(float* MatA, float* MatB, float* MatC, int Width) {
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;

  if (Row < Width && Col < Width) {
    float Pvalue = 0;
    for (int k = 0; k < Width; ++k) {
      Pvalue += MatA[Row * Width + k] * MatB[k * Width + Col];
    }
    MatC[Row * Width + Col] = Pvalue;
  }
}

void initialData(float* ip, const int size) {
  int i;
  for (i = 0; i < size; ++i) {
    ip[i] = ((float)rand() / (float)(RAND_MAX));
    // ip[i] = i;
  }
  return;
}

void displayMatrix(float* A, int nx, int ny) {
  int idx;
  for (int i = 0; i < nx; ++i) {
    for (int j = 0; j < ny; ++j) {
      idx = i * ny + j;
      printf(" %f ", A[idx]);
    }
    printf("\n");
  }
  return;
}

int main() {
  int Width = N;
  int nx = Width;
  int ny = Width;

  int nxy = nx * ny;

  int nBytes = nxy * sizeof(float);

  printf("Matrix of size: nx %d ny %d\n", nx, ny);

  float *h_A, *h_B, *h_C, *h_D, *h_E;

  h_A = (float*)malloc(nBytes);
  h_B = (float*)malloc(nBytes);
  h_C = (float*)malloc(nBytes);
  h_D = (float*)malloc(nBytes);
  h_E = (float*)malloc(nBytes);

  initialData(h_A, nxy);
  initialData(h_B, nxy);
  initialData(h_C, nxy);

  float *d_MatA, *d_MatB, *d_MatC, *d_MatD, *d_MatE;
  hipMalloc((void**)&d_MatA, nBytes);
  hipMalloc((void**)&d_MatB, nBytes);
  hipMalloc((void**)&d_MatC, nBytes);
  hipMalloc((void**)&d_MatD, nBytes);
  hipMalloc((void**)&d_MatE, nBytes);

  hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_MatC, h_C, nBytes, hipMemcpyHostToDevice);

  dim3 block(16, 16);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, 1);

  MatrixMul<<<grid, block>>>(d_MatA, d_MatB, d_MatD, Width);
  hipDeviceSynchronize();

  MatrixMul<<<grid, block>>>(d_MatD, d_MatC, d_MatE, Width);
  hipDeviceSynchronize();

  hipMemcpy(h_D, d_MatD, nBytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_E, d_MatE, nBytes, hipMemcpyDeviceToHost);

  //   printf("Matrix A is \n");
  //   displayMatrix(h_A, nx, ny);

  //   printf("Matrix B is \n");
  //   displayMatrix(h_B, nx, ny);

  //   printf("Matrix C is \n");
  //   displayMatrix(h_C, nx, ny);

  //   printf("Product of Matrix A, B and C is \n");
  //   displayMatrix(h_E, nx, ny);

  hipFree(d_MatA);
  hipFree(d_MatB);
  hipFree(d_MatC);
  hipFree(d_MatD);
  hipFree(d_MatE);

  free(h_A);
  free(h_B);
  free(h_C);
  free(h_D);
  free(h_E);

  hipDeviceReset();

  return 0;
}