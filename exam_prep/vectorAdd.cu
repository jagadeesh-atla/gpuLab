#include <hip/hip_runtime.h>

#include <iostream>

#define N 10

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

__global__ void VectorAdd(int* a, int* b, int* c) {
  int i = threadIdx.x;
  if (i < N) c[i] = a[i] + b[i];
}

int main() {
  int* a = new int[N];
  int* b = new int[N];
  int* c = new int[N];

  int *dev_a, *dev_b, *dev_c;
  hipMalloc(&dev_a, N * sizeof(int));
  hipMalloc(&dev_b, N * sizeof(int));
  hipMalloc(&dev_c, N * sizeof(int));

  for (int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  CHECK_CUDA_ERROR(
      hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(
      hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  VectorAdd<<<1, N>>>(dev_a, dev_b, dev_c);

  CHECK_CUDA_ERROR(
      hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; i++) printf("%d + %d = %d\n", a[i], b[i], c[i]);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  delete[] a;
  delete[] b;
  delete[] c;

  return 0;
}
