#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);

  if (deviceCount == 0)
    printf("There are no devices.\n");
  else
    printf("Detected %d devices.\n", deviceCount);

  int dev = 0, driverVersion = 0, runtimeVersion = 0;
  hipSetDevice(dev);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Device %d: \"%s\"\n", dev, deviceProp.name);

  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  printf("%d.%d\n", driverVersion, runtimeVersion);

  return 0;
}
