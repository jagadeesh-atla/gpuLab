/* 21JE0192 - A JAGADEESH */
/**
 * Sum two matrices with 1D
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

void initialData(float *ip, const int size) {
  int i;

  for (i = 0; i < size; ++i) {
    ip[i] = i;
  }

  return;
}

void displayMatrix(float *A, int nx, int ny) {
  int idx;

  for (int i = 0; i < nx; ++i) {
    for (int j = 0; j < ny; ++j) {
      idx = i * ny + j;
      printf(" %f ", A[idx]);
    }
    printf("\n");
  }

  return;
}

// grid 1D block 1D
__global__ void sumMatrix(float *MatA, float *MatB, float *MatC, int nx,
                          int ny) {
  unsigned int ix = threadIdx.x + (blockIdx.x * blockDim.x);

  if (ix < nx) {
    for (int iy = 0; iy < ny; ++iy) {
      int idx = iy * nx + ix;
      MatC[idx] = MatA[idx] + MatB[idx];
    }
  }
}

int main() {
  int nx = 4;
  int ny = 5;

  int nxy = nx * ny;

  int nBytes = nxy * sizeof(float);

  printf("Matrix size: nx %d ny %d\n", nx, ny);

  float *h_A, *h_B, *h_C;

  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  h_C = (float *)malloc(nBytes);

  initialData(h_A, nxy);
  initialData(h_B, nxy);

  float *d_MatA, *d_MatB, *d_MatC;

  hipMalloc((void **)&d_MatA, nBytes);
  hipMalloc((void **)&d_MatB, nBytes);
  hipMalloc((void **)&d_MatC, nBytes);

  hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

  int dimx = 32;
  dim3 block(dimx, 1);
  dim3 grid((nx + block.x - 1) / block.x, 1);

  sumMatrix<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);

  hipDeviceSynchronize();

  hipMemcpy(h_C, d_MatC, nBytes, hipMemcpyDeviceToHost);

  displayMatrix(h_C, nx, ny);

  hipFree(d_MatA);
  hipFree(d_MatB);
  hipFree(d_MatC);

  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
