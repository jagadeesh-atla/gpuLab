#include <hip/hip_runtime.h>

#include <iostream>

#define N (1 << 16)
#define BLK_SIZE 1 << 6
#define A_BLOCK_STRIDE 32
#define A_T_BLOCK_STRIDE 32

__global__ void transpose(float *A_t, float *A, int a_width, int a_height) {
  int base_idx_a = blockIdx.x * BLK_SIZE + blockIdx.y * A_BLOCK_STRIDE;
  int base_idx_a_t = blockIdx.y * BLK_SIZE + blockIdx.x * A_T_BLOCK_STRIDE;

  int glob_idx_a = base_idx_a + threadIdx.x + a_width * threadIdx.y;
  int glob_idx_a_t = base_idx_a_t + threadIdx.x + a_height * threadIdx.y;

  __shared__ float A_shared[BLK_SIZE][BLK_SIZE + 1];

  A_shared[threadIdx.y][threadIdx.x] = A[glob_idx_a];

  __syncthreads();

  A_t[glob_idx_a_t] = A_shared[threadIdx.x][threadIdx.y];
}
