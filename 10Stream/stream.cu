#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)
#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess) {                              \
      fprintf(stderr, "Error: %s.%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code %d, reason: %s\n", error,        \
              hipGetErrorString(error));                    \
      exit(1);                                               \
    }                                                        \
  }

__global__ void kernel(int *a, int *b, int *c) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N) {
    c[idx] = (a[idx] + b[idx]) / 2.0;
  }
}

int main() {
  hipDeviceProp_t prop;
  int which;
  CHECK(hipGetDevice(&which));
  CHECK(hipGetDeviceProperties(&prop, which));

  if (!prop.deviceOverlap) {
    printf("Device will not handle overlap, so speed up from stream");
    return 0;
  }

  hipEvent_t start, stop;
  float elapsedTime;

  hipStream_t stream;
  int *host_a, *host_b, *host_c;
  int *dev_a, *dev_b, *dev_c;

  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));

  CHECK(hipStreamCreate(&stream));

  CHECK(hipMalloc((void **)&dev_a, N * sizeof(int)));
  CHECK(hipMalloc((void **)&dev_b, N * sizeof(int)));
  CHECK(hipMalloc((void **)&dev_c, N * sizeof(int)));

  CHECK(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));
  CHECK(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));
  CHECK(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int),
                      hipHostMallocDefault));

  for (int i = 0; i < FULL_DATA_SIZE; ++i) {
    host_a[i] = rand();
    host_b[i] = rand();
  }

  CHECK(hipEventRecord(start, 0));
  for (int i = 0; i < FULL_DATA_SIZE; i += N) {
    CHECK(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int),
                          hipMemcpyHostToDevice, stream));
    CHECK(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int),
                          hipMemcpyHostToDevice, stream));

    kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

    CHECK(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int),
                          hipMemcpyDeviceToHost, stream));
  }

  CHECK(hipStreamSynchronize(stream));

  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));

  CHECK(hipEventElapsedTime(&elapsedTime, start, stop));

  printf("Stream ID: %d, \nElapsed Time: %8.6f\n", stream, elapsedTime);

  CHECK(hipHostFree(host_a));
  CHECK(hipHostFree(host_b));
  CHECK(hipHostFree(host_c));

  CHECK(hipFree(dev_a));
  CHECK(hipFree(dev_b));
  CHECK(hipFree(dev_c));

  CHECK(hipStreamDestroy(stream));

  return 0;
}
