#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N 1024

__global__ void distanceGPU(double *a, double *b, double *c) {
    int i=blockIdx.x;
    if (i<N) {
        c[i] = (a[i]-b[i])*(a[i]-b[i]);
    }
}

__global__ void euclideanNormGPU(double *a, double *c) {
    int i=blockIdx.x;
    if (i<N) {
        c[i] = a[i]*a[i];
    }
}

int main(int argc, char **argv) {
    double a[N], b[N], c[N], d;

    double *dev_a, *dev_b, *dev_c;
    hipMalloc((void**) &dev_a, N*sizeof(double));
    hipMalloc((void**) &dev_b, N*sizeof(double));
    hipMalloc((void**) &dev_c, N*sizeof(double));

    for (int i=1; i<=N; i++){
        a[i-1] = i*i;
        b[i-1] = 2*i + 1;
    }

    hipMemcpy(dev_a, a, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(double), hipMemcpyHostToDevice);

    distanceGPU<<<N,1>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, N*sizeof(double), hipMemcpyDeviceToHost);
    d = 0;
    for (int i=0; i<N; i++) {
        d += c[i];
        // printf("%d ", c[i]);
    }

    printf("Distance = %f\n", sqrt(d));

    euclideanNormGPU<<<N,1>>>(dev_a, dev_c);
    hipMemcpy(c, dev_c, N*sizeof(double), hipMemcpyDeviceToHost);
    d = 0;
    for (int i=0; i<N; i++) {
        d += c[i];
        // printf("%d ", c[i]);
    }
    printf("Norm a = %f\n", sqrt(d));

    euclideanNormGPU<<<N,1>>>(dev_b, dev_c);
    hipMemcpy(c, dev_c, N*sizeof(double), hipMemcpyDeviceToHost);
    d = 0;
    for (int i=0; i<N; i++) {
        d += c[i];
        //printf("%d ", c[i]);
    }
    printf("Norm b = %f\n", sqrt(d));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
