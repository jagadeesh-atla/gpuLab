/* 21JE0192 - A JAGADEESH */
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {

        int deviceCount = 0;
        hipGetDeviceCount(&deviceCount);

        if (deviceCount == 0) {
                printf("GPU not available\n");
        } else {
                printf("Detected %d device(s)\n", deviceCount);
        }

        int dev = 0;
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("Warp Size: %d\n", deviceProp.warpSize);
        printf("Max Threads per Multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("Max Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("Max sizes of each dimension of a block: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("Max sizes of each dimension of a grid: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("Max Memory Pitch: %d bytes\n", deviceProp.memPitch);
        return 0;
}
