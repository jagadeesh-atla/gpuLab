#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

void initializeMatrix(float* A, int m, int n){
	for(int i=0; i<m*n; i++)
		A[i] = i;
}

void displayMatrix(float* A, int m, int n) {
	for (int i=0; i<m; i++) {
		for (int j=0; j<n; j++)
			printf("%.0f ", A[i*n + j]);
		printf("\n");
	}
}

__global__ void sumMatrixKernel(float* A, float* B, float* C, int nx, int ny) {
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;

	if ((i<nx) && (j<ny))
		C[i*ny + j] = A[i*ny + j] + B[i*ny + j];
}

int main() {
	int m=16;
	int n=16;
	int size = m*n;

	float *h_A, *h_B, *h_C;
	h_A = (float*)malloc(size*sizeof(float));
	h_B = (float*)malloc(size*sizeof(float));
	h_C = (float*)malloc(size*sizeof(float));

	initializeMatrix(h_A, m, n);
	initializeMatrix(h_B, m, n);	

	float *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, size*sizeof(float));
	hipMalloc((void**)&d_B, size*sizeof(float));
	hipMalloc((void**)&d_C, size*sizeof(float));

	hipMemcpy(d_A, h_A, size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 grid(2, 2, 1);
	dim3 block(8, 8, 1);
	
	sumMatrixKernel<<<grid, block>>>(d_A, d_B, d_C, m, n);
	hipMemcpy(h_C, d_C, size*sizeof(float), hipMemcpyDeviceToHost);
	
	displayMatrix(h_C, m, n);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
