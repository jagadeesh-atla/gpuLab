/* 21JE0192 - A JAGADEESH */
/**
 * Display grid, block and thread details for a block of size (256,3,1)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void blockdetails(void) {
  printf("threadIdx:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z);

  printf("blockIdx:(%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z);

  printf("blockDim:(%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);

  printf("gridDim:(%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
}

int main(int argc, char **argv) {
  int nElem = 3;

  dim3 block(256, 3, 1);
  dim3 grid((nElem + block.x - 1) / block.x, (nElem + block.y - 1) / block.y,
            (nElem + block.z - 1) / block.z);

  printf("blockDim:(%d, %d, %d)\n", block.x, block.y, block.z);
  printf("gridDim:(%d, %d, %d)\n", grid.x, grid.y, grid.z);

  blockdetails<<<grid, block>>>();
  hipDeviceReset();

  return 0;
}
