/* 21JE0192 - A JAGADEESH */
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100
#define BD 256

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t err = call;                                              \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", err, hipGetErrorString(err)); \
      exit(1);                                                                 \
    }                                                                          \
  }

#define max(a, b) (a > b ? a : b)

__global__ void MaxReduction(float *dev_a, float *dev_d) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int t = threadIdx.x;

  __shared__ float partial[BD];
  partial[t] = dev_a[i];

  for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
    __syncthreads();
    if (t % (2 * stride) == 0)
      partial[t] = max(partial[t + stride], partial[t]);
  }
  dev_d[0] = partial[0];
}

int main() {
  float a[N], b[N];
  float *dev_a, *dev_d;

  int bdimx = BD;
  float elapsedTime;

  dim3 block(bdimx);
  dim3 grid((N + block.x - 1) / block.x, 1, 1);

  hipEvent_t start, stop;

  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));

  printf("Array Size is %d\n", N);

  CHECK(hipMalloc((void **)&dev_a, N * sizeof(float)));
  CHECK(hipMalloc((void **)&dev_d, N * sizeof(float)));

  for (int i = 0; i < N; ++i) {
    if (i == 0) {
      a[i] = RAND_MAX;
      continue;
    }
    float x = ((float)rand()) / (float)(RAND_MAX);
    a[i] = x;
    if (N < 25) printf("%f ", x);
  }
  printf("\n");

  CHECK(hipEventRecord(start, 0));
  hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time for Memcpy: %8.6f ms\n", elapsedTime);

  CHECK(hipEventRecord(start, 0));
  MaxReduction<<<grid, block>>>(dev_a, dev_d);
  CHECK(hipMemcpy(b, dev_d, N * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time for Max: %8.6f ms\n", elapsedTime);

  printf("Max = %f\n", b[0]);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(dev_a);
  hipFree(dev_d);

  return 0;
}
