/* 21JE0192 - A JAGADEESH */
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10
#define BD 256

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t err = call;                                              \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", err, hipGetErrorString(err)); \
      exit(1);                                                                 \
    }                                                                          \
  }

#define min(a, b) (a < b ? a : b)

__global__ void MinReduction(float *dev_a, float *dev_d) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int t = threadIdx.x;

  __shared__ float partial[BD];
  partial[t] = dev_a[i];

  for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
    __syncthreads();
    if (t % (2 * stride) == 0 && t < N)
      partial[t] = min(partial[t + stride], partial[t]);
  }
  dev_d[0] = partial[0];
}

int main() {
  float a[N], b[N];
  float *dev_a, *dev_d;

  int bdimx = BD;
  float elapsedTime;

  dim3 block(bdimx);
  dim3 grid((N + block.x - 1) / block.x, 1, 1);

  hipEvent_t start, stop;

  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));

  printf("Array Size is %d\n", N);

  CHECK(hipMalloc((void **)&dev_a, N * sizeof(float)));
  CHECK(hipMalloc((void **)&dev_d, N * sizeof(float)));

  for (int i = 0; i < N; ++i) {
    float x = ((float)rand()) / (float)(RAND_MAX);
    a[i] = x;
    if (N < 25) printf("%f ", x);
  }
  printf("\n");

  CHECK(hipEventRecord(start, 0));
  hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time for Memcpy: %8.6f ms\n", elapsedTime);

  CHECK(hipEventRecord(start, 0));
  MinReduction<<<grid, block>>>(dev_a, dev_d);
  CHECK(hipMemcpy(b, dev_d, N * sizeof(float), hipMemcpyDeviceToHost));
  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time for Min: %8.6f ms\n", elapsedTime);

  printf("Min = %f\n", b[0]);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(dev_a);
  hipFree(dev_d);

  return 0;
}
