#include <hip/hip_runtime.h>

#include <stdio.h>

#define N 100
#define BD 256

#define CHECK(call) \
{ \
	const hipError_t err = call;\
	if (err != hipSuccess) {\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", err,\
				hipGetErrorString(err));\
		exit(1);\
	}\
}

__global__ void SumReduction(float *dev_a, float *dev_d) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int t = threadIdx.x;
	
	__shared__ float partialSum[BD];
	partialSum[t] = dev_a[i];

	for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
		__syncthreads();
		if (t % (2 * stride) == 0 ) 
			partialSum[t] += partialSum[t+stride];
	}
	dev_d[0] = partialSum[0];
}

int main() {
	float a[N], b[N];
	float *dev_a, *dev_d;

	int bdimx = BD;
	float elapsedTime;

	dim3 block(bdimx);
	dim3 grid((N + block.x - 1)/block.x, 1, 1);

	hipEvent_t start, stop;
	
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));
	
	printf("Array Size is %d\n", N);
	
	CHECK(hipMalloc((void **)&dev_a, N * sizeof(float)));
	CHECK(hipMalloc((void **)&dev_d, N * sizeof(float)));
	
	for (int i = 0; i < N; ++i) {	
		float x  = ((float)rand())/(float)(RAND_MAX);
		a[i] = x;
		if (N < 25) printf("%f ", x);
	}
	printf("\n");	

	CHECK(hipEventRecord(start, 0));
	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time for Memcpy: %8.6f ms\n", elapsedTime);

	CHECK(hipEventRecord(start, 0));
	SumReduction<<<grid, block>>>(dev_a, dev_d);
	CHECK(hipMemcpy(b, dev_d, N * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time for Sum: %8.6f ms\n", elapsedTime);

	printf("Sum = %f\n", b[0]);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dev_a);
	hipFree(dev_d);

	return 0;
}

