/**
 * distance b/w two vectors x = i^2, y = (2i + 1) and n = 1024. Also find
 * euclidean norms
 *
 */

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#define N 1024

__global__ void distance(double *a, double *b, double *c) {
  int i = threadIdx.x;
  if (i < N) c[i] = (a[i] - b[i]) * (a[i] - b[i]);
}

__global__ void euclideanNorm(double *a, double *c) {
  int i = threadIdx.x;
  if (i < N) c[i] = a[i] * a[i];
}

int main() {
  double x[N], y[N], temp[N], dist = 0, xNorm = 0, yNorm = 0;

  for (int i = 1; i <= N; ++i) {
    x[i - 1] = i * i;
    y[i - 1] = 2 * i + 1;
  }

  double *dev_x, *dev_y, *dev_d, *dev_xN, *dev_yN;

  hipMalloc((void **)&dev_x, N * sizeof(double));
  hipMalloc((void **)&dev_y, N * sizeof(double));
  hipMalloc((void **)&dev_d, N * sizeof(double));
  hipMalloc((void **)&dev_xN, N * sizeof(double));
  hipMalloc((void **)&dev_yN, N * sizeof(double));

  hipMemcpy(dev_x, x, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, N * sizeof(double), hipMemcpyHostToDevice);

  distance<<<1, N>>>(dev_x, dev_y, dev_d);

  hipMemcpy(temp, dev_d, N * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; ++i) dist += temp[i];

  euclideanNorm<<<1, N>>>(dev_x, dev_xN);
  hipMemcpy(temp, dev_xN, N * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; ++i) xNorm += temp[i];

  euclideanNorm<<<1, N>>>(dev_y, dev_yN);
  hipMemcpy(temp, dev_yN, N * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; ++i) yNorm += temp[i];

  printf(
      "Distance: %.2lf\n"
      "x Norm: %.2lf\n"
      "y Norm: %.2lf\n",
      sqrt(dist), sqrt(xNorm), sqrt(yNorm));

  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_d);
  hipFree(dev_xN);
  hipFree(dev_yN);

  return 0;
}
