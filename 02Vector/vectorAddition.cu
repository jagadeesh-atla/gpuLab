/* 21JE0192 - A JAGADEESH */
/**
 * Element wise sum of vector
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void vecAddGPU(int *a, int *b, int *c) {
  int tid = threadIdx.x;

  if (tid < N) c[tid] = a[tid] + b[tid];
}

int main() {
  int a[N], b[N], c[N];

  int *dev_a, *dev_b, *dev_c;

  hipMalloc((void **)&dev_a, N * sizeof(int));
  hipMalloc((void **)&dev_b, N * sizeof(int));
  hipMalloc((void **)&dev_c, N * sizeof(int));

  for (int i = 0; i < N; ++i) {
    a[i] = -i;
    b[i] = i * i;
  }

  hipError_t err =
      hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    printf("%s\n", hipGetErrorString(err));
    exit(1);
  }

  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  vecAddGPU<<<1, N>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; ++i) printf("%d + %d = %d\n", a[i], b[i], c[i]);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
